#include "hip/hip_runtime.h"
#include "relu.h"

__global__
void kReLuForward(float *a, int sizeX, int sizeY, float* b) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < sizeX && y < sizeY) {
        if (a[y*sizeX + x] < 0.0) {
            b[y*sizeX + x] = 0;
        } else {
            b[y*sizeX + x] = a[y*sizeX + x];
        }
    }
}

__global__
void kReLuBackward(float *a, int sizeX, int sizeY, float* b) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x < sizeX && y < sizeY) {
        if (a[y*sizeX + x] < 0.0) {
            b[y*sizeX + x] = 0;
        } else {
            b[y*sizeX + x] = a[y*sizeX + x];  // TODO: Shouldn't it be 1?
        }
    }
}

ReLuLayer::ReLuLayer(int inputOutput) {
    this->input = this->output = inputOutput;

    // TODO: Remove this by making ReLu Layer an Activation Layer
    this->weights = new Tensor2D(0, 0, (float*)NULL);
    this->bias = new Tensor2D(0, 0, (float*)NULL);
    this->deltaWeights = new Tensor2D(0, 0, (float*)NULL);
    this->deltaBias = new Tensor2D(0, 0, (float*)NULL);
}

Tensor2D* ReLuLayer::forward(Tensor2D* data) {
    // TODO: Check if I really need to create new Tensor here
    float* output;
    hipMalloc((void **)&(output), data->sizeX*data->sizeY*sizeof(float));

    dim3 threadsPerBlock(16, 16);  // TODO: Extract this somewhere else, so we'll be able to easily change it during experiments
    dim3 numBlocks((data->sizeX + threadsPerBlock.x)/threadsPerBlock.x,
                   (data->sizeY + threadsPerBlock.y)/threadsPerBlock.y);
    kReLuForward<<<numBlocks, threadsPerBlock>>>(data->getDeviceData(), data->sizeX, data->sizeY, output);

    return new Tensor2D(data->sizeX, data->sizeY, output);
}
 
Tensor2D* ReLuLayer::backward(Tensor2D* gradients) {
    // TODO: Check if I really need to create new Tensor here
    float* output;
    hipMalloc((void **)&(output), gradients->sizeX*gradients->sizeY*sizeof(float));

    dim3 threadsPerBlock(16, 16);  // TODO: Extract this somewhere else, so we'll be able to easily change it during experiments
    dim3 numBlocks((gradients->sizeX + threadsPerBlock.x)/threadsPerBlock.x,
                   (gradients->sizeY + threadsPerBlock.y)/threadsPerBlock.y);
    kReLuBackward<<<numBlocks, threadsPerBlock>>>(gradients->getDeviceData(), gradients->sizeX, gradients->sizeY, output);

    return new Tensor2D(gradients->sizeX, gradients->sizeY, output);
}
