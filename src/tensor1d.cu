#include "hip/hip_runtime.h"
#include "tensor1d.h"

__global__
void kAdd(float *a, float *b, int N) {
    int i = blockIdx.x;
    if (i < N) {
        a[i] += b[i];
    }
}

Tensor1D::Tensor1D(int size, float* hostData) {
    this->size = size;
    hipMalloc((void **)&(this->devData), this->size*sizeof(float));
    hipMemcpy(devData, hostData, this->size*sizeof(float), hipMemcpyHostToDevice);
}

Tensor1D::~Tensor1D() {
    hipFree(this->devData);
}

float* Tensor1D::getDeviceData() {
    return this->devData;
}

float* Tensor1D::fetchDataFromDevice() {
    float* hostData = (float*)malloc(this->size*sizeof(float));
    hipMemcpy(hostData, this->devData, this->size*sizeof(float), hipMemcpyDeviceToHost);
    return hostData;
}

void Tensor1D::add(Tensor1D* tensor) {
    kAdd<<<this->size, 1>>>(this->getDeviceData(), tensor->getDeviceData(), this->size);
}
