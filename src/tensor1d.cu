#include "hip/hip_runtime.h"
#include "tensor1d.h"

__global__
void kAdd(int *a, int *b, int N) {
    int i = blockIdx.x;
    if (i < N) {
        a[i] += b[i];
    }
}

Tensor1D::Tensor1D(int size, int* hostData) {
    this->size = size;
    hipMalloc((void **)&(this->devData), this->size*sizeof(int));
    hipMemcpy(devData, hostData, this->size*sizeof(int), hipMemcpyHostToDevice);
}

Tensor1D::~Tensor1D() {
    hipFree(this->devData);
}

int* Tensor1D::getDeviceData() {
    return this->devData;
}

int* Tensor1D::fetchDataFromDevice() {
    int* hostData = (int*)malloc(this->size*sizeof(int));
    hipMemcpy(hostData, this->devData, this->size*sizeof(int), hipMemcpyDeviceToHost);
    return hostData;
}

void Tensor1D::add(Tensor1D* tensor) {
    kAdd<<<this->size, 1>>>(this->getDeviceData(), tensor->getDeviceData(), this->size);
}
